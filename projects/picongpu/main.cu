/* general includes needed in param files
 * for lazy quantity definitions:
 *     constexpr quantity<length_d, T=PHYS_UNITS_REP_TYPE>
 */
//define PHYS_UNITS_REP_TYPE float
#include "phys/units/quantity.hpp"
// for printing
#include "phys/units/quantity_io.hpp"

/* constants */
#include "phys/units/physical_constants.hpp"

/* CT math */

/* used namespaces */
using namespace phys::units;
using namespace phys::units::literals;

/* define more literals for plasma physics */
namespace phys
{
namespace units
{
namespace literals
{
    // more literals
    QUANTITY_DEFINE_LITERALS( V, electric_potential_d )
    QUANTITY_DEFINE_LITERALS( T, magnetic_flux_density_d )
    //QUANTITY_DEFINE_LITERALS( K, thermodynamic_temperature_d )
    QUANTITY_DEFINE_SCALING_LITERALS( qe, electric_charge_d, -1.602176462e-19 )
    QUANTITY_DEFINE_SCALING_LITERALS( qp, electric_charge_d, 1.602176462e-19 )
    QUANTITY_DEFINE_SCALING_LITERALS( me, mass_d, 9.10938356e-31 )
    QUANTITY_DEFINE_SCALING_LITERALS( mp, mass_d, 1.672621898e-27 )
    QUANTITY_DEFINE_SCALING_LITERALS( eV, energy_d, 1.60217733e-19 )
}
}
}

/* input helper file(s) */
#include "solver_helper.hpp"

/* param file(s) */
#include "param.hpp"

/* PIConGPU unit system */
namespace picRatioSI
{
    constexpr quantity< length_d > length{
        // SI::DELTA_T * TYPICAL_SPEED (c)
        Iteration::t * c
    };
    constexpr quantity< mass_d > mass{
        // SI::BASE_MASS (* typical weighting)
        detail::magnitude_tag, 9.10938356e-31
    };
    constexpr quantity< time_interval_d > time_interval{
        // SI::DELTA_T
        Iteration::t
    };
    constexpr quantity< electric_current_d > electric_current{
        // SI::BASE_CHARGE (* typical weighting) / UNIT_TIME
        e / time_interval
    };
    constexpr quantity< thermodynamic_temperature_d > thermodynamic_temperature{
        detail::magnitude_tag, 1.0
    };
    constexpr quantity< amount_of_substance_d > amount_of_substance{
        detail::magnitude_tag, 1.0
    };
    constexpr quantity< luminous_intensity_d > luminous_intensity{
        detail::magnitude_tag, 1.0
    };
}

/* to PIConGPU unit system */
template< typename T_Dimensions >
struct ToPIC;

template<>
template< int D1, int D2, int D3, int D4, int D5, int D6, int D7 >
struct ToPIC< dimensions< D1, D2, D3, D4, D5, D6, D7 > >
{
    using dim = dimensions< D1, D2, D3, D4, D5, D6, D7 >;
    
    template<
        typename DX,
        typename X
    >
    constexpr auto
    operator()( quantity< DX, X > const & x ) const
    -> X //detail::Quotient<dim, DX, dim, X>
    {
        /*static_assert(
            std::is_same<DX, dim>::value,
            "Dimension mismatch in variable ... macro magic..."
        );*/
        return x /
            (
                nth_power< D1 >( picRatioSI::length ) *
                nth_power< D2 >( picRatioSI::mass ) *
                nth_power< D3 >( picRatioSI::time_interval ) *
                nth_power< D4 >( picRatioSI::electric_current ) *
                nth_power< D5 >( picRatioSI::thermodynamic_temperature ) *
                nth_power< D6 >( picRatioSI::amount_of_substance ) *
                nth_power< D7 >( picRatioSI::luminous_intensity )
            );
    }
};

#include <iostream>
//include <hip/hip_runtime.h>


/* a host funtion using the input */
void
printStuff()
{
    using namespace phys::units::io;
    
    constexpr auto ddd = Iteration::t;
    std::cout << ddd << std::endl;

    constexpr auto two_e = e + e + e - 1.0_qe;
    
    std::cout << ToPIC< electric_charge_d >()( two_e ) << std::endl;
    // must fail:
    //std::cout << ToPIC< length_d >()( two_e ) << std::endl;
}
/* a kernel using the input */
/*
__device__ void
dev_foo()
{
    constexpr auto ddd = B::t;
    printf("%d", ddd.to( second ) );
}*/


int main()
{
    printStuff();
    
    return 0;
}

